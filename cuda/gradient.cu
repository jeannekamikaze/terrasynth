#include "hip/hip_runtime.h"
#include "gradient.h"
#include <cstdio>
#include <cstdlib>
#include <cmath>

#define TABLE_SIZE 256
#define TABLE_MASK (TABLE_SIZE-1)

typedef unsigned char uchar;

// Device Code

inline __device__ float dot (const float2& a, const float2& b)
{
    return a.x*b.x + a.y*b.y;
}

// 3t^2 - 2t^3
inline __device__ float scurve (float t)
{
    return t * t * (t * -2 + 3);
}

// 6t^5 - 15t^4 + 10t^3
inline __device__ float scurve5 (float t)
{
    return t * t * t * (t * (t * 6 - 15) + 10);
}

inline __device__ float lerp (float a, float b, float t)
{
    return a + (b-a)*t;
}

#define PERM(x)      perms [(x) & TABLE_MASK]
#define INDEX(ix,iy) PERM ((ix)+PERM((iy)))

inline __device__ float2 operator- (const float2& a, const float2& b)
{
    return make_float2 (a.x-b.x, a.y-b.y);
}

__device__ float noise (const uchar* perms, const float2* grads, float x, float y)
{
    int x0 = x;
    int y0 = y;
    int x1 = x0 + 1;
    int y1 = y0 + 1;

    float2 p  = make_float2 (x,y);
    float2 p0 = make_float2 (x0,y0);
    float2 p1 = make_float2 (x1,y0);
    float2 p2 = make_float2 (x0,y1);
    float2 p3 = make_float2 (x1,y1);

    float2 g0 = grads[INDEX(x0,y0)];
    float2 g1 = grads[INDEX(x1,y0)];
    float2 g2 = grads[INDEX(x0,y1)];
    float2 g3 = grads[INDEX(x1,y1)];

    float s = dot (g0, p-p0);
    float t = dot (g1, p-p1);
    float u = dot (g2, p-p2);
    float v = dot (g3, p-p3);

    float sx = scurve5 (x - (float)x0);
    float sy = scurve5 (y - (float)y0);
    float a = lerp (s, t, sx);
    float b = lerp (u, v, sx);
    float c = lerp (a, b, sy);

    // Map from [-1,1] to [0,1]
    c = c*0.5f + 0.5f;

    return c;
}

__global__ void kernel_noise (const uchar* perms, const float2* grads, float s, int w, int h, float* image)
{
    unsigned n = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned x = n % w;
    unsigned y = n / w;
    float* p = image + n;
    *p = noise (perms, grads, (float)x*s, (float)y*s);
}

__global__ void kernel_fbm
(const uchar* perms, const float2* grads, float octaves, float lacunarity, float gain, float fbm_max, float s, int w, int h, float* image)
{
    unsigned n = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned x = n % w;
    unsigned y = n / w;
    float* p = image + n;

    *p = 0.0f;
    float f = 1.0f;
    float a = 1.0f;
    for (int n = 0; n < octaves; ++n) {
        *p += a * noise (perms, grads, (float)x*f*s, (float)y*f*s);
        f *= lacunarity;
        a *= gain;
    }
    *p /= fbm_max;
}

// Host Code

uchar  h_perms [TABLE_SIZE];
float2 h_grads [TABLE_SIZE];

uchar*  d_perms;
float2* d_grads;

void setup (int seed)
{
    srand (seed);

    // Construct a random permutaion table of values uniformly distributed
    // in the 0..255 range.
    for (int i = 0; i < TABLE_SIZE; ++i) h_perms[i] = i;
    for (int i = 0; i < TABLE_SIZE; ++i) {
        uchar j = rand ();
        h_perms[i] ^= h_perms[j] ^= h_perms[i];
    }

    // Construct a random gradient table of values uniformly distributed
    // along the unit circle.
    float step = 2*M_PI / (float) TABLE_SIZE;
    float angle = 0.0f;
    for (int i = 0; i < TABLE_SIZE; ++i, angle += step) {
        h_grads[i].x = cos (angle);
        h_grads[i].y = sin (angle);
    }

    // Copy to device memory.
    hipMalloc ((void**)&d_perms, TABLE_SIZE);
    hipMemcpy (d_perms, h_perms, TABLE_SIZE, hipMemcpyHostToDevice);
    hipMalloc ((void**)&d_grads, TABLE_SIZE*sizeof(float2));
    hipMemcpy (d_grads, h_grads, TABLE_SIZE*sizeof(float2), hipMemcpyHostToDevice);
}

void clean ()
{
    hipFree (d_perms);
    hipFree (d_grads);
}

void noise (int seed, float* h_image, int w, int h, int cell_size, int freq)
{
    setup (seed);

    unsigned n = (unsigned) w * (unsigned) h;
    unsigned nf = n * sizeof(float);
    float* d_image;
    hipMalloc ((void**)&d_image, nf);

    const float s = (float) freq / (float) cell_size;
    unsigned tpb = 1024;
    unsigned nb = n / tpb;
    kernel_noise<<<nb,tpb>>>(d_perms, d_grads, s, w, h, d_image);

    hipMemcpy (h_image, d_image, nf, hipMemcpyDeviceToHost);
    hipFree (h_image);

    clean();
}

float geom (float r, float n)
{
    return (1.0f - pow(r,n)) / (1.0f - r);
}

void fbm (int seed, float* h_image, int w, int h, int cell_size, float lacunarity, float H, int octaves)
{
    setup (seed);

    unsigned n = (unsigned) w * (unsigned) h;
    unsigned nf = n * sizeof(float);
    float* d_image;
    hipMalloc ((void**)&d_image, nf);

    const float s = 1.0f / (float) cell_size;
    const float gain = pow (lacunarity, -2*H);
    const float fbm_max = gain == 1.0f ? 1.0f : geom (gain, octaves);
    unsigned tpb = 1024;
    unsigned nb = n / tpb;
    kernel_fbm<<<nb,tpb>>>(d_perms, d_grads, octaves, lacunarity, gain, fbm_max, s, w, h, d_image);

    hipMemcpy (h_image, d_image, nf, hipMemcpyDeviceToHost);
    hipFree (h_image);

    clean();
}
